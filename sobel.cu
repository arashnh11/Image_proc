#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <fcntl.h>
#include <unistd.h>
#include "string.h"
#include "hip/hip_runtime.h"
#include <helper_image.h>     // helper for image and data comparison

// This code is written by Arash Nemati Hayati 00763261 - Feb 26, 2016
#define opt 1 // The version to be used

#define DEFAULT_THRESHOLD  4000
#define DEFAULT_FILENAME "BWstop-sign.ppm"

#define BLOCK_SIZE 32 // Number of threads in x and y direction - Maximum Number of threads per block = 32 * 32 = 1024
#define TILE_SIZE 32 // Tile size for memory hierarchy optimizations
#define version_1 1 // Using global memory
#define version_2 2 // Using memory hierarchy optimizations
#define N_runs 2 // Number of Runs - Exclude the first run timing.

__global__ void sobel( int xd_size, int yd_size, int maxdval, int d_thresh, unsigned int *input , int *output)
{
	
// Version 1 - Using global memory
	
	if (opt == 1){	
        int magnitude, sum1, sum2;
        int i = blockIdx.y * blockDim.y + threadIdx.y; // Row index
        int j = blockIdx.x * blockDim.x + threadIdx.x; // Column index
	
        if ((i < yd_size) && (j < xd_size))
        {
        output[i * xd_size + j] = 0;
        }
	__syncthreads();

        if ((i > 0) && (i < yd_size - 1) && (j > 0) && (j < xd_size - 1))
        {
        int offset = i * xd_size + j; 
	
        sum1 =  input[ xd_size * (i-1) + j+1 ] -     input[ xd_size*(i-1) + j-1 ]
        + 2 * input[ xd_size * (i)   + j+1 ] - 2 * input[ xd_size*(i)   + j-1 ]
        +     input[ xd_size * (i+1) + j+1 ] -     input[ xd_size*(i+1) + j-1 ];

        sum2 = input[ xd_size * (i-1) + j-1 ] + 2 * input[ xd_size * (i-1) + j ]  + input[ xd_size * (i-1) + j+1 ]
            - input[xd_size * (i+1) + j-1 ] - 2 * input[ xd_size * (i+1) + j ] - input[ xd_size * (i+1) + j+1 ];

        magnitude =  sum1*sum1 + sum2*sum2;

      if (magnitude > d_thresh){
        output[offset] = 255;
        }
      else{
        output[offset] = 0;
        }
    }
	__syncthreads();
}

// Version 2 - Using memory hierarchy optimization
	
	if (opt == 2){
	int magnitude, sum1, sum2;
	int tile_width = TILE_SIZE;
	__shared__ int temp[(TILE_SIZE) * (TILE_SIZE)];
	
	int i = blockIdx.y * blockDim.y + threadIdx.y; // row
        int j = blockIdx.x * blockDim.x + threadIdx.x; // column
	int tid_x = threadIdx.y; // x index analogy to i
        int tid_y = threadIdx.x; // y index analogy to j
	
	// Initialize the output
        if ((i < yd_size) && (j < xd_size))
        {
        output[i * xd_size + j] = 0;
        }
        __syncthreads();
	
	// Initialize the temp tile matrix
	temp[tid_x * tile_width + tid_y]  = 0; // Initializae the temp tile matrix which stores a block chunk of data
	__syncthreads();
	
	// Load data from global memory to shared memory
	temp[tid_x * tile_width + tid_y] = input[i * (xd_size) + j];
	__syncthreads();
	
	// No image processing on the image boundaries
	if (i > 0 && j > 0 && i < yd_size - 1 && j < xd_size - 1){	
	
	// Loop over the tile and apply the filter
	if ((tid_x > 0) && (tid_x < tile_width - 1)  && (tid_y > 0) && (tid_y < tile_width - 1))
        {
        int offset = i * xd_size + j;
	
	int tem_iminus1jplus1 = temp[ tile_width * (tid_x-1) + tid_y+1];
        int tem_iminus1jminus1 = temp[ tile_width * (tid_x-1) + tid_y-1 ];
        int tem_iplus1jplus1 = temp[ tile_width * (tid_x+1) + tid_y+1];

        sum1 =  tem_iminus1jplus1 -     tem_iminus1jminus1
        + 2 * temp[ tile_width * (tid_x)   + tid_y+1 ] - 2 * temp[ tile_width*(tid_x)   + tid_y-1 ]
        +     tem_iplus1jplus1 -     temp[ tile_width*(tid_x+1) + tid_y-1 ];

        sum2 = tem_iminus1jminus1 + 2 * temp[ tile_width * (tid_x-1) + tid_y ]  
        + tem_iminus1jplus1 - temp[tile_width * (tid_x+1) + tid_y-1 ] 
        - 2 * temp[ tile_width * (tid_x+1) + tid_y ] - tem_iplus1jplus1;

/*
        sum1 =  temp[ tile_width * (tid_x-1) + tid_y+1 ] -     temp[ tile_width*(tid_x-1) + tid_y-1 ]
        + 2 * temp[ tile_width * (tid_x)   + tid_y+1 ] - 2 * temp[ tile_width*(tid_x)   + tid_y-1 ]
        +     temp[ tile_width * (tid_x+1) + tid_y+1 ] -     temp[ tile_width*(tid_x+1) + tid_y-1 ];

        sum2 = temp[ tile_width * (tid_x-1) + tid_y-1 ] + 2 * temp[ tile_width * (tid_x-1) + tid_y ]  
	+ temp[ tile_width * (tid_x-1) + tid_y+1 ] - temp[tile_width * (tid_x+1) + tid_y-1 ] 
	- 2 * temp[ tile_width * (tid_x+1) + tid_y ] - temp[ tile_width * (tid_x+1) + tid_y+1 ];
*/	
	magnitude = sum1 * sum1 + sum2 * sum2;
	int e_ig = 0;
	if (magnitude > d_thresh){
        //output[offset] = 255;
	e_ig = 255;
        }
/*        else{
        //output[offset] = 0;
	eig = 0;
        }*/
	output[offset] = e_ig;
	}
	__syncthreads();

	// For the boundary elements on the tile use the global memory	
	if ((i == blockIdx.y * blockDim.y + blockDim.y - 1) || (j == blockIdx.x * blockDim.x + blockDim.x - 1) || 
	   (i == blockIdx.y * blockDim.y) || (j == blockIdx.x * blockDim.x))
	{
	int offset = i * xd_size + j;
	int inp_iminus1jplus1 = input[ xd_size * (i-1) + j+1];
	int inp_iminus1jminus1 = input[ xd_size * (i-1) + j-1 ];
	int inp_iplus1jplus1 = input[ xd_size *	(i+1) + j+1];

        sum1 =  inp_iminus1jplus1 - inp_iminus1jminus1
        + 2 * input[ xd_size * (i)   + j+1 ] - 2 * input[ xd_size*(i)   + j-1 ]
        +     inp_iplus1jplus1 -     input[ xd_size*(i+1) + j-1 ];

        sum2 = inp_iminus1jminus1 + 2 * input[ xd_size * (i-1) + j ]  + inp_iminus1jplus1
            - input[xd_size * (i+1) + j-1 ] - 2 * input[ xd_size * (i+1) + j ] - inp_iplus1jplus1;


/*	sum1 =  input[ xd_size * (i-1) + j+1 ] -     input[ xd_size*(i-1) + j-1 ]
        + 2 * input[ xd_size * (i)   + j+1 ] - 2 * input[ xd_size*(i)   + j-1 ]
        +     input[ xd_size * (i+1) + j+1 ] -     input[ xd_size*(i+1) + j-1 ];

        sum2 = input[ xd_size * (i-1) + j-1 ] + 2 * input[ xd_size * (i-1) + j ]  + input[ xd_size * (i-1) + j+1 ]
            - input[xd_size * (i+1) + j-1 ] - 2 * input[ xd_size * (i+1) + j ] - input[ xd_size * (i+1) + j+1 ]; 
*/	
        magnitude =  sum1*sum1 + sum2*sum2;
	int e_ig = 0;
      if (magnitude > d_thresh){
       // output[offset] = 255;
	e_ig = 255;
        }
    /*  else{
      //  output[offset] = 0;
	e_ig = 0;
        }*/
	output[offset] = e_ig;
	}
	__syncthreads();
	}
    }	
}

unsigned int *read_ppm( char *filename, int & xsize, int & ysize, int & maxval ){
  
  if ( !filename || filename[0] == '\0') {
    fprintf(stderr, "read_ppm but no file name\n");
    return NULL;  // fail
  }

  fprintf(stderr, "read_ppm( %s )\n", filename);
  int fd = open( filename, O_RDONLY);
  if (fd == -1) 
    {
      fprintf(stderr, "read_ppm()    ERROR  file '%s' cannot be opened for reading\n", filename);
      return NULL; // fail 

    }

  char chars[1024];
  int num = read(fd, chars, 1000);

  if (chars[0] != 'P' || chars[1] != '6') 
    {
      fprintf(stderr, "Texture::Texture()    ERROR  file '%s' does not start with \"P6\"  I am expecting a binary PPM file\n", filename);
      return NULL;
    }

  unsigned int width, height, maxvalue;


  char *ptr = chars+3; // P 6 newline
  if (*ptr == '#') // comment line! 
    {
      ptr = 1 + strstr(ptr, "\n");
    }

  num = sscanf(ptr, "%d\n%d\n%d",  &width, &height, &maxvalue);
  fprintf(stderr, "read %d things   width %d  height %d  maxval %d\n", num, width, height, maxvalue);  
  xsize = width;
  ysize = height;
  maxval = maxvalue;
  
  unsigned int *pic = (unsigned int *)malloc( width * height * sizeof(unsigned int));
  if (!pic) {
    fprintf(stderr, "read_ppm()  unable to allocate %d x %d unsigned ints for the picture\n", width, height);
    return NULL; // fail but return
  }

  // allocate buffer to read the rest of the file into
  int bufsize =  3 * width * height * sizeof(unsigned char);
  if (maxval > 255) bufsize *= 2;
  unsigned char *buf = (unsigned char *)malloc( bufsize );
  if (!buf) {
    fprintf(stderr, "read_ppm()  unable to allocate %d bytes of read buffer\n", bufsize);
    return NULL; // fail but return
  }





  // TODO really read
  char duh[80];
  char *line = chars;

  // find the start of the pixel data.   no doubt stupid
  sprintf(duh, "%d\0", xsize);
  line = strstr(line, duh);
  //fprintf(stderr, "%s found at offset %d\n", duh, line-chars);
  line += strlen(duh) + 1;

  sprintf(duh, "%d\0", ysize);
  line = strstr(line, duh);
  //fprintf(stderr, "%s found at offset %d\n", duh, line-chars);
  line += strlen(duh) + 1;

  sprintf(duh, "%d\0", maxval);
  line = strstr(line, duh);


  fprintf(stderr, "%s found at offset %d\n", duh, line - chars);
  line += strlen(duh) + 1;

  long offset = line - chars;
  lseek(fd, offset, SEEK_SET); // move to the correct offset
  long numread = read(fd, buf, bufsize);
  fprintf(stderr, "Texture %s   read %ld of %ld bytes\n", filename, numread, bufsize); 

  close(fd);


  int pixels = xsize * ysize;
  for (int i=0; i<pixels; i++) pic[i] = (int) buf[3*i];  // red channel

 

  return pic; // success
}

void write_ppm( char *filename, int xsize, int ysize, int maxval, int *pic) 
{
  FILE *fp;
  
  fp = fopen(filename, "w");
  if (!fp) 
    {
      fprintf(stderr, "FAILED TO OPEN FILE '%s' for writing\n");
      exit(-1); 
    }
  int x,y;
  
  
  fprintf(fp, "P6\n"); 
  fprintf(fp,"%d %d\n%d\n", xsize, ysize, maxval);
  
  int numpix = xsize * ysize;
  for (int i=0; i<numpix; i++) {
    unsigned char uc = (unsigned char) pic[i];
    fprintf(fp, "%c%c%c", uc, uc, uc); 
  }
  fclose(fp);

}
	
main( int argc, char **argv )
{

  int thresh = DEFAULT_THRESHOLD;
  char *filename;
  hipError_t error;
  filename = strdup( DEFAULT_FILENAME);
  
  if (argc > 1) {
    if (argc == 3)  { // filename AND threshold
      filename = strdup( argv[1]);
       thresh = atoi( argv[2] );
    }
    if (argc == 2) { // default file but specified threshhold
      
      thresh = atoi( argv[1] );
    }

    fprintf(stderr, "file %s    threshold %d\n", filename, thresh); 
  }


  int xsize, ysize, maxval;
 
  unsigned int *pic = read_ppm( filename, xsize, ysize, maxval ); 
  unsigned int *dev_pic;

  int numbytes =  xsize * ysize * 1 * sizeof( int );
  int *result_gpu = (int *) malloc( numbytes );
  int *result_cpu = (int *) malloc( numbytes );
 
if (!result_cpu) {
    fprintf(stderr, "sobel() unable to malloc %d bytes\n", numbytes);
    exit(-1); // fail
  }
if (!result_gpu) {
    fprintf(stderr, "sobel() unable to malloc %d bytes\n", numbytes);
    exit(-1); // fail
  }

//***********CPU implementation*************************************************
  int i, j, magnitude, sum1, sum2;
  int *out = result_cpu;

  for (int col=0; col<ysize; col++) {
    for (int row=0; row<xsize; row++) {
      *out++ = 0;
    }
  }

  for (i = 1;  i < ysize - 1; i++) {
    for (j = 1; j < xsize -1; j++) {

      int offset = i*xsize + j;

      sum1 =  pic[ xsize * (i-1) + j+1 ] -     pic[ xsize*(i-1) + j-1 ]
        + 2 * pic[ xsize * (i)   + j+1 ] - 2 * pic[ xsize*(i)   + j-1 ]
        +     pic[ xsize * (i+1) + j+1 ] -     pic[ xsize*(i+1) + j-1 ];

      sum2 = pic[ xsize * (i-1) + j-1 ] + 2 * pic[ xsize * (i-1) + j ]  + pic[ xsize * (i-1) + j+1 ]
            - pic[xsize * (i+1) + j-1 ] - 2 * pic[ xsize * (i+1) + j ] - pic[ xsize * (i+1) + j+1 ];

      magnitude =  sum1*sum1 + sum2*sum2;

      if (magnitude > thresh)
        result_cpu[offset] = 255;
      else
        result_cpu[offset] = 0;
    }
  }

write_ppm( "result_cpu.ppm", xsize, ysize, 255, result_cpu);


//***********GPU kernel implementation**********************************************
  int *dev_result;
 
  hipMalloc ((void **)&dev_result, numbytes);
  hipMalloc ((void **)&dev_pic, numbytes);
  error = hipMalloc ((void **) &dev_pic, numbytes);

    if (error != hipSuccess)
    {
        printf("hipMemcpy (devic,d) returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }
  hipMemcpy( dev_pic, pic, numbytes, hipMemcpyHostToDevice);

int gridsize_x = xsize/BLOCK_SIZE + 1;
int gridsize_y = ysize/BLOCK_SIZE + 1;

        dim3 dimgrid(gridsize_x, gridsize_y, 1); // The grid has #gridsize blocks in x and 1 block in y and 1 block in z direction
        dim3 dimblock(BLOCK_SIZE, BLOCK_SIZE, 1);

// Block/thread decompositions report

        fprintf(stderr,".....Number of block in x dir......%d\n",gridsize_x);
        fprintf(stderr,".....Number of block in y dir......%d\n",gridsize_y);
	fprintf(stderr,".....Total Number of blocks........%d\n",gridsize_x * gridsize_y);
        fprintf(stderr,".....Number of threads in x dir....%d\n",BLOCK_SIZE);
        fprintf(stderr,".....Number of threads in y dir....%d\n",BLOCK_SIZE);
	fprintf(stderr,".....Total Number of threads........%d\n",BLOCK_SIZE * BLOCK_SIZE);
//        fprintf(stderr,".....xsize (Number of columns).....%d\n",xsize);
//        fprintf(stderr,".....ysize (Number of rows)........%d\n",ysize,"\n");

// warmup to avoid timing startup
sobel<<<dimgrid, dimblock>>>(xsize, ysize, maxval, thresh, dev_pic, dev_result);


for (int counter = 1; counter <= N_runs; ++counter){

  // Initialize timer
  hipEvent_t start,stop;
  float elapsed_time;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start,0);

  sobel<<<dimgrid, dimblock>>>(xsize, ysize, maxval, thresh, dev_pic, dev_result);
  
  hipDeviceSynchronize();
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_time,start, stop);
  hipMemcpy( result_gpu, dev_result, numbytes, hipMemcpyDeviceToHost);
  
  write_ppm( "result_gpu.ppm", xsize, ysize, 255, result_gpu);

const char *kernelName;
kernelName = "sobel";
bool success;
success = true;

// Structure: Compare CPU results with GPU results
          bool res = compareData(result_cpu, result_gpu, xsize * ysize * 1, 0.01f, 0.0f);

        if (res == false)
        {
            printf("*** %s kernel FAILED ***\n", kernelName);
            bool success = false;
        }
	else
	{ 
	if (counter == N_runs){
	   printf("\n");
	   printf("*** %s kernel PASSED ***\n", kernelName);
	   printf("The outputs of CPU version and GPU version are identical.\n");
	  fprintf(stderr, "sobel done\n");
	  printf("The operation was successful, time = %2.6f %s\n", elapsed_time, "ms");
	}
	}
}
  hipFree(dev_result);
  free(result_cpu);
  free(result_gpu);
}

